#include "hip/hip_runtime.h"
// nvcc -diag-suppress 550 -O2 -lm grayscott_par_1.c -o grayscott_par_1
// ./grayscott_par_1 256 5000 1 0.16 0.08 0.060 0.062

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// just for testing (saving the final image)
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// initialize U and V grids (2D) with a square in the middle
void initUV(float *U, float *V, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            U[i * n + j] = 1.0f;
            V[i * n + j] = 0.0f;
        }
    }
    int r = n / 8;
    for (int i = n / 2 - r; i < n / 2 + r; i++) {
        for (int j = n / 2 - r; j < n / 2 + r; j++) {
            U[i * n + j] = 0.75f;
            V[i * n + j] = 0.25f;
        }
    }
}

// save float array as an image, just for testing
void save_grayscale_image(const char* filename, float* data, int n) {
    // allocate a buffer for 8-bit pixels
    unsigned char* pixels = (unsigned char*)malloc(n * n * sizeof(unsigned char));
    // convert float [0, 1] to unsigned char [0, 255]
    for (int i = 0; i < n * n; i++) {
        pixels[i] = (unsigned char)(data[i] * 255.0f);
    }
    // save as PNG
    stbi_write_png(filename, n, n, 1, pixels, n);
    free(pixels);
}

__global__ void gray_scott_kernel(float* U, float* V, float* U_new, float* V_new, int n, float dt, float du, float dv, float f, float k) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) {
        // Wrap-around indices using modulus
        int up    = (i - 1 + n) % n;
        int down  = (i + 1) % n;
        int left  = (j - 1 + n) % n;
        int right = (j + 1 + n) % n;

        float center_u = U[i * n + j];
        float center_v = V[i * n + j];

        float laplacian_u = U[up * n + j] + U[down * n + j] + U[i * n + left] + U[i * n + right] - 4.0f * center_u;
        float laplacian_v = V[up * n + j] + V[down * n + j] + V[i * n + left] + V[i * n + right] - 4.0f * center_v;

        float UV_square = center_u * center_v * center_v;

        U_new[i * n + j] = center_u + dt * (-UV_square + f * (1.0f - center_u) + du * laplacian_u);
        V_new[i * n + j] = center_v + dt * ( UV_square - (f + k) * center_v + dv * laplacian_v);
    }
}

// here is where the magic happens, baby
float gray_scott(float* U, float* V, float* U_new, float* V_new, int n, int steps, float dt, float du, float dv, float f, float k) {
    // Device pointers
    float *d_U, *d_V, *d_U_new, *d_V_new;
    int size = n * n * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_U, size);
    hipMalloc(&d_V, size);
    hipMalloc(&d_U_new, size);
    hipMalloc(&d_V_new, size);

    // Copy initial data to device
    hipMemcpy(d_U, U, size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, V, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(32, 32);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    // Use CUDA events to measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timer
    hipEventRecord(start);
    for (int cur_step = 0; cur_step < steps; cur_step++) {
        gray_scott_kernel<<<gridSize, blockSize>>>(d_U, d_V, d_U_new, d_V_new, n, dt, du, dv, f, k);

        // Swap pointers (on device)
        float* tmp;

        tmp    = d_U;
        d_U    = d_U_new;
        d_U_new= tmp;

        tmp    = d_V;
        d_V    = d_V_new;
        d_V_new= tmp;
    }
    // Stop timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Copy result back to host
    hipMemcpy(U, d_U, size, hipMemcpyDeviceToHost);
    hipMemcpy(V, d_V, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_U_new);
    hipFree(d_V_new);

    // Print time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    return milliseconds;
}

int main(int argc, char *argv[]) {

    if (argc < 8) {
        printf("USAGE: grayscott_seq width steps time_step diff_u diff_v feed_factor kill_rate\n");
        exit(EXIT_FAILURE);
    }

    int n = atoi(argv[1]); // grid width
    int steps = atoi(argv[2]); // number of steps of the algorithm
    float dt = atof(argv[3]); // time step size
    float du = atof(argv[4]); // diffusion rate for u
    float dv = atof(argv[5]); // diffusion rate for v
    float f = atof(argv[6]); // feed rate
    float k = atof(argv[7]); // kill rate

    // allocate memory for the grids
    int grid_size = n * n;
    float *U      = (float*) malloc(grid_size * sizeof(float));
    float *V      = (float*) malloc(grid_size * sizeof(float));
    float *U_new  = (float*) malloc(grid_size * sizeof(float));
    float *V_new  = (float*) malloc(grid_size * sizeof(float));

    // initialize U and V
    initUV(U, V, n);

    // gray scott function, where everything happens basically
    float milliseconds = gray_scott(U, V, U_new, V_new, n, steps, dt, du, dv, f, k);

    // optionally visualize the end result (V grid)
    save_grayscale_image("V_end.png", V, n);

    // free resources
    free(U);
    free(V);
    free(U_new);
    free(V_new);

    // print out the time
    printf("Kernel Execution time is: %0.3f seconds \n", milliseconds / 1000.0f);

    return 0;
}